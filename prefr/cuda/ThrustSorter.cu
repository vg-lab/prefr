/*
 * Copyright (c) 2014-2020 VG-Lab/URJC.
 *
 * Authors: Sergio E. Galindo <sergio.galindo@urjc.es>
 *
 * This file is part of PReFr <https://github.com/vg-lab/prefr>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 */

#include <thrust/sort.h>
#include <thrust/copy.h>
#include "ThrustSorter.cuh"

namespace prefr
{

  ThrustSorter::ThrustSorter( )
  { }

  ThrustSorter::~ThrustSorter( )
  { }

  void ThrustSorter::initDistanceArray( ICamera* camera )
  {
    _distances = new CUDADistanceArray( _particles.size, camera );

    CUDADistanceArray* cda = static_cast< CUDADistanceArray* >( _distances );

    cda->deviceID.resize( _particles.size );
    cda->deviceDistances.resize( _particles.size );
  }

  void ThrustSorter::sort( SortOrder order )
  {

    CUDADistanceArray* cda = static_cast< CUDADistanceArray* >( _distances );

    std::vector< int >::iterator hostidbegin = _distances->ids.begin( );
    std::vector< int >::iterator hostidend = hostidbegin + _aliveParticles;

    std::vector< float >::iterator hostdistbegin =
        _distances->distances.begin( );

    std::vector< float >::iterator hostdistend =
        hostdistbegin  + _aliveParticles;

    thrust::device_vector< int >::iterator deviceidbegin =
        cda->deviceID.begin( );

    thrust::device_vector< int >::iterator deviceidend =
        deviceidbegin + _aliveParticles;

    thrust::device_vector< float >::iterator devicedistbegin =
        cda->deviceDistances.begin( );

    thrust::device_vector< float >::iterator devicedistend =
        devicedistbegin + _aliveParticles;

    thrust::sequence( deviceidbegin, deviceidend );
    thrust::copy( hostdistbegin, hostdistend, devicedistbegin );

    if( order == SortOrder::Ascending )
      thrust::sort_by_key( devicedistbegin, devicedistend,
                           deviceidbegin, thrust::less< float >( ));
    else
      thrust::sort_by_key( devicedistbegin, devicedistend,
                           deviceidbegin, thrust::greater< float >( ));

    thrust::copy( deviceidbegin, deviceidend, hostidbegin );

  }

  void ThrustSorter::updateCameraDistance( const glm::vec3& cameraPosition,
                                           bool renderDeadParticles )
  {
    _aliveParticles = 0;
    _distances->resetCounter( );

    for( auto cluster : *_clusters )
    {
      if( cluster->active( ) || renderDeadParticles )
      {
        for( tparticle particle = cluster->particles( ).begin( );
             particle != cluster->particles( ).end( );
             particle++ )
        {
          updateParticleDistance( &particle, cameraPosition,
                                  renderDeadParticles );
          _aliveParticles++;
        }
      }
    }


  }

  void ThrustSorter::updateParticleDistance( const tparticle_ptr current,
                                             const glm::vec3& cameraPosition,
                                             bool renderDeadParticles )
  {
    DistanceUnit* dist = _distances->next( );
    CUDADistanceArray* cda = static_cast< CUDADistanceArray* >( _distances );
    cda->translatedIDs[ _distances->current ] = current->id( );

    ( *_distances ).distances[ _distances->current ] =
        current->alive( )  || renderDeadParticles ?
        length2( current->position( ) - cameraPosition ) :
        -1;

  }

}
